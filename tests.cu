#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <math.h>
#include "kernels.cuh"


// Function to check the results
bool checkResults(float* output, float* expected, int size, float tolerance = 1e-5) {
    for (int i = 0; i < size; i++) {
        if (std::abs(output[i] - expected[i]) > tolerance) {
            return false;
        }
    }
    return true;
}


int main() {
    // Example parameters
    int width = 7;  // Width of the input matrix
    int height = 7; // Height of the input matrix
    int filterWidth = 3;
    int inputChannels = 1;
    int outputChannels = 1;
    int imageSize = width * height * inputChannels;
    int outputWidth = width - filterWidth + 1;
    int outputHeight = height - filterWidth + 1;
    int outputSize = outputWidth * outputHeight * outputChannels;
    int filterSize = filterWidth * filterWidth * inputChannels * outputChannels;

    //pooling
    int outputWidth_pooling = floor(width/2);
    int outputHeight_pooling = floor(height/2);

    // Allocate host memory and initialize test data
    
    //Dense

    float weights[9] = {
        1,1,2,
        2,1,3,
        1,4,2
    };

    
    float input[3] = {3,1,2};
    float e_out_dense[3] = {6,13,11};
    float out_dense[3] = {0,0,0};

    //Conv
    
    float h_input[49] = {
        0,1,1,1,0,0,0,
        0,0,1,1,1,0,0,
        0,0,0,1,1,1,0,
        0,0,0,1,1,0,0,
        0,0,1,1,0,0,0,
        0,1,1,0,0,0,0,
        1,1,0,0,0,0,0
    };

    float h_filter[9] = {
        1,0,1,
        0,1,0,
        1,0,1
    };

    float h_output_same[imageSize];
    float h_output[outputSize];
    float h_output_pooling[outputWidth_pooling*outputHeight_pooling*inputChannels];

    float h_expectedOutput[25] = {
        1,4,3,4,1,
        1,2,4,3,3,
        1,2,3,4,1,
        1,3,3,1,1,
        3,3,1,1,0
    };

    float h_expectedOutput_pooling[outputWidth_pooling*outputHeight_pooling*inputChannels] = {
        0.25, 1, 0.25,
        0, 0.5, 0.75,
        0.25, 0.75,0
    };

    float h_expectedOutput_same[49] = {
        0, 2, 2, 3, 1, 1, 0,
        1, 1, 4, 3, 4, 1, 1,
        0, 1, 2, 4, 3, 3, 0,
        0, 1, 2, 3, 4, 1, 1,
        1, 1, 3, 3, 1, 1, 0,
        1, 3, 3, 1, 1, 0, 0,
        2, 2, 1, 1, 0, 0, 0
    };

    // Allocate device memory
    float *d_input, *d_output, *d_filter, *d_output_pooling,*d_output_same;
    float *input_l, *weights_l, *out_dense_l;

    hipMalloc(&input_l, 3 * sizeof(float));
    hipMalloc(&weights_l, 9 * sizeof(float));
    hipMalloc(&out_dense_l, 3 * sizeof(float));

    hipMalloc(&d_input, imageSize * sizeof(float));
    hipMalloc(&d_output, outputSize * sizeof(float));
    hipMalloc(&d_output_same, imageSize * sizeof(float));
    hipMalloc(&d_filter, filterSize * sizeof(float));
    hipMalloc(&d_output_pooling, outputWidth_pooling*outputHeight_pooling*inputChannels * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, h_input, imageSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, filterSize * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(weights_l, weights, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(input_l, input, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_dense_l, out_dense, 3 * sizeof(float), hipMemcpyHostToDevice);
    
    // Set up kernel launch parameters and run the kernel

    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((outputWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, 
               (outputHeight + threadsPerBlock.y - 1) / threadsPerBlock.y, 
               outputChannels);
    //convolution3D<<<numBlocks, threadsPerBlock>>>(d_input, inputChannels, width, filterWidth, 1, d_filter, d_output, false);           
    conv2DKernelCombined<<<numBlocks, threadsPerBlock>>>(d_input, d_output, d_filter, width, height, filterWidth, inputChannels, outputChannels);

    dim3 threadsPerBlock_same(16, 16, 1);
    dim3 numBlocks_same((width + threadsPerBlock_same.x - 1) / threadsPerBlock_same.x, 
               (height + threadsPerBlock_same.y - 1) / threadsPerBlock_same.y, 
               outputChannels);

    //convolution3D<<<numBlocks, threadsPerBlock>>>(d_input, inputChannels, width, filterWidth, 1, d_filter, d_output_same, true);
    conv2DKernelCombined<<<numBlocks_same, threadsPerBlock_same>>>(d_input, d_output_same, d_filter, width, height, filterWidth, inputChannels, outputChannels,true);


    dim3 threadsPerBlock_pool(16, 16, 1);
    dim3 numBlocks_pool((outputWidth_pooling + threadsPerBlock_pool.x - 1) / threadsPerBlock_pool.x, 
               (outputHeight_pooling + threadsPerBlock_pool.y - 1) / threadsPerBlock_pool.y, 
               inputChannels);

    averagePoolingKernel<<<numBlocks_pool, threadsPerBlock_pool>>>(d_input, d_output_pooling, width, height, outputWidth_pooling, outputHeight_pooling, inputChannels, 2);


    // Copy result back to host
    hipMemcpy(h_output_same, d_output_same, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_output, d_output, outputSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_pooling, d_output_pooling, outputWidth_pooling*outputHeight_pooling*inputChannels * sizeof(float), hipMemcpyDeviceToHost);

    // After copying the output back to the host
    std::cout << "Kernel output:" << std::endl;
    for (int i = 0; i < outputSize; ++i) {  
        std::cout << h_output[i] << " ";
        if ((i + 1) % outputWidth == 0) std::cout << std::endl;
    }
    std::cout << "Expected output Conv:" << std::endl;
    for (int i = 0; i < 25; ++i) {  
        std::cout << h_expectedOutput[i] << " ";
        if ((i + 1) % 5 == 0) std::cout << std::endl;
    }

    // Check the results
    if (checkResults(h_output, h_expectedOutput, outputSize)) {
        std::cout << "Test Passed!" << std::endl;
    } else {
        std::cout << "Test Failed!" << std::endl;
    }

    std::cout << "Kernel output Pooling:" << std::endl;
    for (int i = 0; i < outputWidth_pooling*outputHeight_pooling*inputChannels; ++i) {  
        std::cout << h_output_pooling[i] << " ";
        if ((i + 1) % outputWidth_pooling == 0) std::cout << std::endl;
    }

    std::cout << "Expected output Pooling:" << std::endl;
    for (int i = 0; i < outputWidth_pooling*outputHeight_pooling*inputChannels; ++i) {  
        std::cout << h_expectedOutput_pooling[i] << " ";
        if ((i + 1) % outputWidth_pooling == 0) std::cout << std::endl;
    }

      if (checkResults(h_output_pooling, h_expectedOutput_pooling, outputWidth_pooling*outputHeight_pooling*inputChannels)) {
        std::cout << "Test Passed!" << std::endl;
    } else {
        std::cout << "Test Failed!" << std::endl;
    }

    std::cout << "Kernel output same:" << std::endl;
    for (int i = 0; i < imageSize; ++i) {  
        std::cout << h_output_same[i] << " ";
        if ((i + 1) % width == 0) std::cout << std::endl;
    }

    if (checkResults(h_output_same, h_expectedOutput_same, imageSize)) {
        std::cout << "Test Passed!" << std::endl;
    } else {
        std::cout << "Test Failed!" << std::endl;
    }

    dim3 threadsPerBlock_d1(16);  
    dim3 numBlocks_d1((3 + threadsPerBlock_d1.x - 1) / threadsPerBlock_d1.x);

    denseLayerKernel<<<numBlocks_d1, threadsPerBlock_d1>>>(input_l, out_dense_l, weights_l,nullptr, 3, 3, false);
    hipDeviceSynchronize();
    hipMemcpy(out_dense, out_dense_l, 3 * sizeof(float), hipMemcpyDeviceToHost);

     std::cout << "output dense:" << std::endl;
    for (int i = 0; i < 3; ++i) {  
        std::cout << out_dense[i] << " ";
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);

    return 0;
}
